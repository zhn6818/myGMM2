#include "hip/hip_runtime.h"
#include "comman.h"


__global__ void initKernel(uchar* img, int step, int w, int h, float* nodeP, int nodeStep, double cov)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
	
    if (x >= w || y >= h)
        return;
    // if(y == 0 && x == 0)
    // {
    //     printf("hello");
    // }
    uchar b = img[y * step + x * 3];
    uchar g = img[y * step + x * 3 + 1];
    uchar r = img[y * step + x * 3 + 2];
    // printf("%d, %d, %d, %d, %d \n", x, y, b, g, r);
    
    int indexNode = x + y * w;
    char* data = (char*)nodeP + indexNode * nodeStep;

    NodePixelGpu *nodeDev = (NodePixelGpu *)(data);
    nodeDev->realSize = 0;
    int index = nodeDev->realSize;
    
    nodeDev->gaussian[index].mean[0] = b;
    nodeDev->gaussian[index].mean[1] = g;
    nodeDev->gaussian[index].mean[2] = r;
    nodeDev->gaussian[index].covariance = cov;
    nodeDev->gaussian[index].weight = 1.0;
    nodeDev->realSize = nodeDev->realSize + 1;
    // if(y == 500 && x == 500)
    // {
    //     printf("Dev realSize： %f \n", nodeDev->realSize);
    //     printf("%d, %d, %f, %f, %f %f\n", x, y, (float)nodeDev->gaussian[0].mean[0], (float)nodeDev->gaussian[0].mean[1], (float)nodeDev->gaussian[0].mean[2], nodeDev->realSize);
    //     printf("%d, %d %d \n", (int)b, (int)g, (int)r);
    // }
}


void InitNode(cv::cuda::GpuMat &tmpImg, float* nodeP, double cov)
{
    std::cout << "tmpImg: " << tmpImg.cols << " " << tmpImg.rows << std::endl;

    const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(tmpImg.cols, blockDim.x), iDivUp(tmpImg.rows,blockDim.y));
    initKernel<<<gridDim, blockDim>>>(tmpImg.ptr<uchar>(), tmpImg.step, tmpImg.cols, tmpImg.rows, nodeP, sizeof(NodePixelGpu), cov);

    hipDeviceSynchronize();

    std::cout << std::endl;
}


__global__ void processKernel(uchar* out, int outStep, uchar* img, int step, int w, int h, float* nodeP, int nodeStep, double cov, double alpha, double alpha_bar, double prune, double cfbar)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
	
    if (x >= w || y >= h)
        return;

    double rVal = img[y * step + x * 3];
    double gVal = img[y * step + x * 3 + 1];
    double bVal = img[y * step + x * 3 + 2];

    double sum = 0.0;
    bool close = false;
    int background = 255;
    float weight = 0.0;
    double mal_dist = 0.0;
    double mult = 0.0;
    double temp_cov;
    double var;
    double muR, muG, muB, dR, dG, dB;

    int indexNode = x + y * w;
    char* data = (char*)nodeP + indexNode * nodeStep;
    NodePixelGpu *nodeDev = (NodePixelGpu *)(data);

    // printf("%d, %d, %f, %f, %f \n", x, y, rVal, gVal, bVal);
    // if(y == 500 && x == 500)
    // {
    //     printf("%d, %d, %f, %f, %f %f\n", x, y, nodeDev->gaussian[0].mean[0], nodeDev->gaussian[0].mean[1], nodeDev->gaussian[0].mean[2], nodeDev->realSize);
    // }
    // if(x == 0 && y == 0)
    // {
        if(nodeDev->realSize > MaxSize)
        {
            nodeDev->realSize = nodeDev->realSize - 1;
            nodeDev->gaussian[MaxSize].mean[0] = 0;
            nodeDev->gaussian[MaxSize].mean[1] = 0;
            nodeDev->gaussian[MaxSize].mean[2] = 0;
            nodeDev->gaussian[MaxSize].covariance = 0;
            nodeDev->gaussian[MaxSize].weight = 0;
        }
        for(int k = 0; k < nodeDev->realSize; k++)
        {
            weight = nodeDev->gaussian[k].weight;
            mult = alpha / weight;
            weight = weight * alpha_bar + prune;
            // printf("weight: %f", weight);
            if(close == false)
            {
                muR = nodeDev->gaussian[k].mean[0];
                muG = nodeDev->gaussian[k].mean[1];
                muB = nodeDev->gaussian[k].mean[2];
                dR = rVal - muR;
                dG = gVal - muG;
                dB = bVal - muB;
                var = nodeDev->gaussian[k].covariance;
                mal_dist = (dR * dR + dG * dG + dB * dB);
                if ((sum < cfbar) && (mal_dist < 16.0 * var * var))
                {
                    background = 0;
                }
                if (mal_dist < 9.0 * var * var)
                {

                    weight += alpha;
                    close = true;
                    nodeDev->gaussian[k].mean[0] = muR + mult * dR;
                    nodeDev->gaussian[k].mean[1] = muG + mult * dG;
                    nodeDev->gaussian[k].mean[2] = muB + mult * dB;
                    temp_cov = var + mult * (mal_dist - var);
                    nodeDev->gaussian[k].covariance = temp_cov < 5.0 ? 5.0 : (temp_cov > 20.0 ? 20.0 : temp_cov);
                }

            }
            if(weight < -prune)
            {
                nodeDev->realSize = nodeDev->realSize - 1;
                nodeDev->gaussian[k].mean[0] = 0;
                nodeDev->gaussian[k].mean[1] = 0;
                nodeDev->gaussian[k].mean[2] = 0;
                nodeDev->gaussian[k].covariance = 0;
                nodeDev->gaussian[k].weight = 0;
            }else{
                sum += weight;
                nodeDev->gaussian[k].weight = weight;
            }
        }
        if (close == false)
        {
            int index = nodeDev->realSize;
            nodeDev->gaussian[index].mean[0] = rVal;
            nodeDev->gaussian[index].mean[1] = gVal;
            nodeDev->gaussian[index].mean[2] = bVal;
            nodeDev->gaussian[index].covariance = cov;
            nodeDev->gaussian[index].weight = alpha;
            nodeDev->realSize = nodeDev->realSize + 1;
        }
        for (int m = 0; m < nodeDev->realSize; m++)
        {
            nodeDev->gaussian[m].weight /= sum;
        }
        // printf("back %d, ", background);
        for (int m = nodeDev->realSize - 1; m > 0 && (m - 1) >= 0; m--)
        {
            if(nodeDev->gaussian[m].weight > nodeDev->gaussian[m - 1].weight)
            {
                // printf("%f, %f, %f \n", *((float*)nodeDev->gaussian[m]), *((float*)nodeDev->gaussian[m] + 1), *((float*)nodeDev->gaussian[m] + 2));
                float t1 = nodeDev->gaussian[m].mean[0];
                float t2 = nodeDev->gaussian[m].mean[1];
                float t3 = nodeDev->gaussian[m].mean[2];
                float t4 = nodeDev->gaussian[m].covariance;
                float t5 = nodeDev->gaussian[m].weight;
                nodeDev->gaussian[m].mean[0] = nodeDev->gaussian[m - 1].mean[0];
                nodeDev->gaussian[m].mean[1] = nodeDev->gaussian[m - 1].mean[1];
                nodeDev->gaussian[m].mean[2] = nodeDev->gaussian[m - 1].mean[2];
                nodeDev->gaussian[m].covariance = nodeDev->gaussian[m - 1].covariance;
                nodeDev->gaussian[m].weight = nodeDev->gaussian[m - 1].weight;
                nodeDev->gaussian[m - 1].mean[0] = t1;
                nodeDev->gaussian[m - 1].mean[1] = t2;
                nodeDev->gaussian[m - 1].mean[2] = t3;
                nodeDev->gaussian[m - 1].covariance = t4;
                nodeDev->gaussian[m - 1].weight = t5;
            }
        }

        out[y * outStep + x] = (uchar)background;
    // }

}



void processNode(cv::cuda::GpuMat &tmpImg, cv::cuda::GpuMat &outImg, float *nodeP, double cov, double alpha, double alpha_bar, double prune, double cfbar)
{
    assert(tmpImg.cols == outImg.cols && tmpImg.rows == outImg.rows);
    // std::cout << std::endl;
    const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(tmpImg.cols, blockDim.x), iDivUp(tmpImg.rows,blockDim.y));
    processKernel<<<gridDim, blockDim>>>(outImg.ptr<uchar>(), outImg.step, tmpImg.ptr<uchar>(), tmpImg.step, tmpImg.cols, tmpImg.rows, nodeP, sizeof(NodePixelGpu), cov, alpha, alpha_bar, prune, cfbar);

    hipDeviceSynchronize();

    // std::cout << std::endl;
}

__global__ void getImgKernel(uchar* imgGmm, int step, int h, int w, float* nodeP, int nodeStep)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
	
    if (x >= w || y >= h)
        return;

    int indexNode = x + y * w;
    char* data = (char*)nodeP + indexNode * nodeStep;
    NodePixelGpu *nodeDev = (NodePixelGpu *)(data);

    float b = nodeDev->gaussian[0].mean[0];
    float g = nodeDev->gaussian[0].mean[1];
    float r = nodeDev->gaussian[0].mean[2];


    imgGmm[y * step + x * 3] = static_cast<uchar>(b);
    imgGmm[y * step + x * 3 + 1] = static_cast<uchar>(g);
    imgGmm[y * step + x * 3 + 2] = static_cast<uchar>(r);

    // if(y == 450 && x == 650)
    // {
    //     printf("ttttttttt%d, %d, %f, %f, %f %f\n", x, y, nodeDev->gaussian[0].mean[0], nodeDev->gaussian[0].mean[1], nodeDev->gaussian[0].mean[2], nodeDev->realSize);
    //     printf("%d %d %d \n", (int)imgGmm[y * step + x * 3], (int)imgGmm[y * step + x * 3 + 1], (int)imgGmm[y * step + x * 3 + 2]);
    // }
}


void GetNode(cv::cuda::GpuMat &imgGmm, float *nodeP)
{
    // assert(imgGmm.cols == outImg.cols && imgGmm.rows == outImg.rows);
    const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(imgGmm.cols, blockDim.x), iDivUp(imgGmm.rows,blockDim.y));
    // std::cout << "imgGmm: " << imgGmm.cols << " " << imgGmm.rows << " " << std::endl;
    getImgKernel<<<gridDim, blockDim>>>(imgGmm.ptr<uchar>(), imgGmm.step, imgGmm.rows, imgGmm.cols, nodeP, sizeof(NodePixelGpu));
    hipDeviceSynchronize();
}

__global__ void processDiffKernel(uchar* img1, int img1step, int w, int h, uchar* img2, int img2step, uchar* src, int srcstep, uchar* result, int resultstep)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= w || y >= h)
        return;

    int b = img1[y * img1step + x * 3];
    int g = img1[y * img1step + x * 3 + 1];
    int r = img1[y * img1step + x * 3 + 2];

    int b1 = img2[y * img2step + x * 3];
    int g1 = img2[y * img2step + x * 3 + 1];
    int r1 = img2[y * img2step + x * 3 + 2];

    int diffb = abs(b1 - b);
    int diffg = abs(g1 - g);
    int diffr = abs(r1 - r);

    if(sqrt(float(diffb * diffb + diffr * diffr + diffg * diffg)) > 200)
    {
        result[y * resultstep + x * 3] = 255;
        result[y * resultstep + x * 3 + 1] = 255;
        result[y * resultstep + x * 3 + 2] = 255;
    }
}

void processDiff(cv::cuda::GpuMat &img1, cv::cuda::GpuMat& img2, cv::cuda::GpuMat& src, cv::cuda::GpuMat& result)
{
    assert(img1.cols == img2.cols && img1.rows == img2.rows);
    const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(img1.cols, blockDim.x), iDivUp(img1.rows,blockDim.y));
    processDiffKernel<<<gridDim, blockDim>>>(img1.ptr<uchar>(), img1.step, img1.cols, img1.rows, img2.ptr<uchar>(), img2.step, src.ptr<uchar>(), src.step, result.ptr<uchar>(), result.step);
    hipDeviceSynchronize();
}

__global__ void caculateSim(uchar* img1, int img1step, uchar* img2, int img2step, float* result, int resizeStep, int w, int h, int binSize)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= w || y >= h)
        return;
    float sum = 0;
    float sum1 = 0;
    float sum2 = 0;
    for(int i = x * binSize; i < x * binSize + binSize; i ++)
    {
        for(int j = y * binSize; j < y * binSize + binSize; j++)
        {
            int b = img1[j * img1step + i * 3];
            int g = img1[j * img1step + i * 3 + 1];
            int r = img1[j * img1step + i * 3 + 2];

            int b1 = img2[j * img2step + i * 3];
            int g1 = img2[j * img2step + i * 3 + 1];
            int r1 = img2[j * img2step + i * 3 + 2];
            // if(x == 20 && y == 20)
            // {
            //     printf("%d, %d, %d, %d \n", i, j, b, b1);
            // }
            sum += (b - b1) * (b - b1);
            sum1 += b * b;
            sum2 += b1 * b1;
        }
    }
    float value = (sum / (std::sqrt(sum1) * std::sqrt(sum2)));
    // if(x == 20 && y == 20)
    // {
    //     printf("%f ,%f \n", sum, value);
    // }
    result[y * resizeStep + x] = value;
}

void caculateSim(cv::Mat &img1, cv::Mat &img2, cv::Mat& result, int binSize)
{
    cv::cuda::GpuMat img1Gpu, img2Gpu, resultGpu;
    img1Gpu.upload(img1);
    img2Gpu.upload(img2);
    resultGpu.upload(result);
    int sizeH = img1.rows / binSize;
    int sizeW = img1.cols / binSize;

    const dim3 blockDim(8, 8);
    const dim3 gridDim(iDivUp(sizeW, blockDim.x), iDivUp(sizeH,blockDim.y)); 

    caculateSim<<<gridDim, blockDim>>>(img1Gpu.ptr<uchar>(), img1Gpu.step, img2Gpu.ptr<uchar>(), img2Gpu.step, resultGpu.ptr<float>(), resultGpu.step / sizeof(float), sizeW, sizeH, binSize);

    resultGpu.download(result);
    std::cout << std::endl;
}

__global__ void diffsquarekernel(float* src, int srcstep, float* filter, int filterstep, float* result, int resizeStep, int src_w, int src_h, int filter_w, int filter_h, int boundary)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    
    if(x < boundary || y < boundary || x + boundary >= src_w || y + boundary >= src_h)
    {
        return;
    }
    float valueSum = 0;
    // if(x == 8 && y==8)
    // {
    //共享内存 方案 行不通
    for(int j = 0; j < filter_h; j++)
    {
        for(int i = 0; i < filter_w; i++)
        {
            int xshift = i - boundary;
            int yshift = j - boundary;

            float srcvalue = src[(y + yshift) * srcstep + (x + xshift)];
            float filtervalue = filter[j * filterstep + i];
            float diff = srcvalue - filtervalue;
            
            valueSum += diff * diff;
        }
    }
        // printf("%f, %f, %f", valueSum, valueSum, valueSum);
    // }

    result[y * resizeStep + x] = valueSum;

}

void diffsquare(cv::cuda::GpuMat & src, cv::cuda::GpuMat & filter, cv::cuda::GpuMat & result)
{
    int sizeH = src.rows;
    int sizeW = src.cols;
    int boundary = filter.rows / 2;
    const dim3 blockDim(8, 8);
    const dim3 gridDim(iDivUp(sizeW, blockDim.x), iDivUp(sizeH,blockDim.y));
    diffsquarekernel<<<gridDim, blockDim>>>(src.ptr<float>(), src.step / sizeof(float), filter.ptr<float>(), filter.step / sizeof(float), result.ptr<float>(), result.step / sizeof(float), src.cols, src.rows, filter.cols, filter.rows, boundary);
    // std::cout << std::endl;
}

__global__ void initValue(A* dev_testA, int m, int n)
{
    printf("%d  %d \n", m, n);

    for(int i = 0; i < m; i++)
    {
        for(int j = 0; j < n; j++)
        {
            // (testA + i)->a[j]  = j;
            dev_testA[i].dev_a[j] = j;
        }
    }
}

void testArray()
{
    std::cout << "testArray start " << std::endl;
    A * testA;
    int m = 10, n = 20;
    testA = (A*)malloc(m * sizeof(A));
    A *dev_testA;
    hipMalloc((void**)&dev_testA, m * sizeof(A));
    for(int i = 0; i < m; i++)
    {
        testA[i].a = (float*)malloc(n * sizeof(float));
        hipMalloc((void**)&testA[i].dev_a, n * sizeof(float));
        hipMemset(testA[i].dev_a, 0, n * sizeof(float));
    }
    
    hipMemcpy(dev_testA, testA, m * sizeof(A), hipMemcpyHostToDevice);
    for(int i = 0; i < m; i++)
    {
        hipMemcpy(testA[i].dev_a, testA[i].a, n * sizeof(float), hipMemcpyHostToDevice);
    }
    initValue<<<1, 1>>>(dev_testA, m, n);

    for(int i = 0; i < m; i++)
    {
        hipMemcpy(testA[i].a, testA[i].dev_a, n * sizeof(float), hipMemcpyDeviceToHost);
    }

    for(int i = 0; i < m; i++)
    {
        for(int j = 0; j < n; j++)
        {
            std::cout << testA[i].a[j] << " ";
        }
        std::cout << std::endl;
    }


    for(int i = 0; i < m; i++)
    {
        free(testA[i].a);
    }
    free(testA);


    std::cout << "testArray end" << std::endl;
}



